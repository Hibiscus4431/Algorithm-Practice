#include "hip/hip_runtime.h"
/*

 Author: Paige Smith
 Date: October 2025
 Project Name: missile_sim.cu
 Description:
    Simulate N independent missiles in 2D (x, y) where each
    missile has position, velocity, mass, drag coefficient,
    and optimal thrust. Skills goal: Use CUDA to parallelize
    the simulation of the missiles.
 Tools: Nvidia GPU, CUDA Toolkit, Python, C++
 This was made with the guidance of OpenAI for mathematical equations and code structure.
*/

// Algorithm design:
/*
to keep things simple use linearized drag force: a_drag = -k * v
where k is the drag coefficient and v is the velocity vector

gravity: a_gravity = (0, -g) where g = 9.81 m/s^2
thrust: optional constant acceleration in some direction a_thrust = (a_tx, a_ty)

equation: v += dt * (a_gravity + a_drag + a_thrust)
          pos += dt * v

*/

// includes libraries
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <fstream>
#include <vector>
#include <iostream>

// struct object to hold missile data
struct Missile
{
    float x, y;     // position
    float vx, vy;   // velocity
    float mass;     // mass
    float drag_k;   // drag coefficient
    float thrust_x; // thrust in x direction
    float thrust_y; // thrust in y direction
};

// GPU Kernel threading function where each thread simulates one missile across all timesteps
__global__ void simulateKernel(Missile *missiles, int N, float dt, int steps, float g)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N)
        return;

    // Local (registers) copy for speed
    Missile m = missiles[idx];

    // Time integration loop
    for (int s = 0; s < steps; ++s)
    {
        // acceleration components
        float ax = -m.drag_k * m.vx + m.thrust_x;     // linear drag
        float ay = -m.drag_k * m.vy + m.thrust_y - g; // linear drag + gravity

        // Euler integration
        m.vx += ax * dt;
        m.vy += ay * dt;
        m.x += m.vx * dt;
        m.y += m.vy * dt;

        // Ground collision
        if (m.y < 0.0f)
        {
            // Stay at ground and zero vertical velocity
            m.y = 0.0f;
            m.vy = 0.0f;
            // keep simulating or break here
        }
    }

    // Write back to global memory
    missiles[idx] = m;
};

int main(int argc, char **argv)
{
    // Simulation parameters (can be parsed)
    const int N = 100000;   // number of missiles
    const float dt = 0.01f; // time step
    const int steps = 500;  // number of time steps
    const float g = 9.81f;  // gravity m/s^2

    // Allocate host missiles
    std::vector<Missile> h_missiles;
    h_missiles.resize(N);

    // Initialize missiles with random data
    for (int i = 0; i < N; ++i)
    {
        float angle = 30.0f * (i % 20) * 2.0f;             // Vary angle
        float speed = 50.0f + (i % 50);                    // Vary speed
        float rad = angle * 3.14159265f / 180.0f;          // degrees to radians
        h_missiles[i].x = 0.0f;                            // start at origin
        h_missiles[i].y = 0.0f;                            // start at ground level
        h_missiles[i].vx = speed * cosf(rad);              // initial velocity x
        h_missiles[i].vy = speed * sinf(rad);              // initial velocity y
        h_missiles[i].mass = 100.0f;                       // mass in kg
        h_missiles[i].drag_k = 0.1f + ((i % 10) * 0.001f); // vary drag coefficient

        // small thrust for a few missiles
        if (i % 1000 == 0)
        {
            h_missiles[i].thrust_x = 0.0f;
            h_missiles[i].thrust_y = 0.5f; // slight upward thrust
        }
        else
        {
            h_missiles[i].thrust_x = 0.0f;
            h_missiles[i].thrust_y = 0.0f; // no thrust
        }
    }

    // allocate device memory
    Missile *d_missiles = nullptr;
    size_t bytes = sizeof(Missile) * N;
    hipError_t cerr = hipMalloc((void **)&d_missiles, bytes);
    if (cerr != hipSuccess)
    {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(cerr));
        return 1;
    }

    //copy host to device
    hipMemcpy(d_missiles, h_missiles.data(), bytes, hipMemcpyHostToDevice);

    // Launch kernel with enough blocks to cover all missiles
    int blockSizr = 256;
    int gridSize = (N + blockSizr - 1) / blockSizr;

   // warmup or hipDeviceSynchronize timing
    simulateKernel<<<gridSize, blockSize>>>(d_missiles, N, dt, steps, g);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_missiles);
        return 1;
    }
    hipDeviceSynchronize();

    // Copy back
    hipMemcpy(h_missiles.data(), d_missiles, bytes, hipMemcpyDeviceToHost);

    //Save results to CSV file (final positions and velocities)
    std::ofstream out("missiles_results.csv");
    out << "id,x,y,vx,vy,mass,drag_k,thrust_x,thrust_y\n";
    for (int i = 0; i < N; ++i)
    {
        auto &m = h_missiles[i];
        out << i << "," << m.x << "," << m.y << "," << m.vx << "," 
        << m.vy << "," << m.mass << "," << m.drag_k << "," 
        << m.thrust_x << "," << m.thrust_y << "\n";
    }
    out.close();

    // Free device memory
    hipFree(d_missiles);
    printf
    return 0;
}
